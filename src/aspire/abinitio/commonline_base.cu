#include "hip/hip_runtime.h"
#include <stdint.h>
#include <cupy/complex.cuh>

extern "C" __global__
void build_clmatrix_kernel(int n, int m, int r, const complex<double>* __restrict__ pf, int16_t* __restrict__ clmatrix, double* __restrict__  cl_dist, double* __restrict__ shifts_1d, int n_shifts, double* __restrict__ shifts,  const complex<double>* __restrict__ shift_phases)
{
  /* n n_img */
  /* m,r st (n, m, r) = pf.shape, ie len(pf[i])  */

  /* thread index (1d), represents "i" index */
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;

  /* no-op when out of bounds */
  if(i >= n) return;
  if(j >= n) return;
  /* no-op lower triangle */
  if(j <= i) return;

  int ind;
  int k;
  int s;
  int cl1, cl2;
  int best_cl1, best_cl2, best_s;
  double dist, best_cl_dist;
  double p1, p2;
  complex<double> pfik, pfjk;

  best_s = -99999;
  best_cl1 = -1;
  best_cl2 = -1;
  best_cl_dist = -1/0;

  for(cl1=0; cl1<m; cl1++){
    for(cl2=0; cl2<m; cl2++){
      for(s=0; s<n_shifts; s++){
        p1 = 0;
        p2 = 0;
        /* inner most dim of dot (matmul) */
        for(k=0; k<r; k++){
          pfik = pf[k*m*n + cl1*n + i];
          pfjk = conj(pf[k*m*n + cl2*n + j]) * shift_phases[s*r + k];
          p1 += real(pfik) * real(pfjk);
          p2 += imag(pfik) * imag(pfjk);
        }

        dist = p1 - p2;
        if(dist > best_cl_dist){
          best_cl_dist = dist;
          best_cl1 = cl1;
          best_cl2 = cl2;
          best_s = s;
        }

        dist = p1 + p2;
        if(dist > best_cl_dist){
          best_cl_dist = dist;
          best_cl1 = cl1;
          best_cl2 = cl2 + m; // m is pf.shape[1], which should be n_theta//2...
          best_s = s;
        }

      } /* s */
    } /* cl2 */
  }/* cl1 */


  /* update global best for i, j*/
  ind = i*n + j;
  clmatrix[ind] = best_cl1;
  clmatrix[j*n+i] = best_cl2;  /* [j,i] */
  cl_dist[ind] = 2*best_cl_dist;  // 2 of mystery
  shifts_1d[ind] = shifts[best_s];

}
