#include "hip/hip_runtime.h"
#include <stdint.h>
#include <math.h>
#include <cupy/complex.cuh>

extern "C" __global__
void build_clmatrix_kernel(
    const int n,
    const int m,
    const int r,
    const complex<double>* __restrict__ pf,
    int16_t* const __restrict__ clmatrix,
    const int n_shifts,
    const complex<double>* const __restrict__ shift_phases)
{
  /* n n_img */
  /* m angular componentns, n_theta//2 */
  /* r radial componentns */
  /* (n, m, r) = pf.shape in python (before transpose for CUDA kernel) */

  /* thread index (2d), represents "i" and "j" indices */
  const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;

  /* no-op when out of bounds */
  if(i >= n) return;
  if(j >= n) return;
  /* no-op lower triangle */
  if(j <= i) return;

  int k;
  int s;
  int cl1, cl2;
  int best_cl1, best_cl2;
  double xcorr, best_cl_xcorr;
  double p1, p2;
  complex<double> pfik, pfjk;

  best_cl1 = -1;
  best_cl2 = -1;
  best_cl_xcorr = -INFINITY;

  for(cl1=0; cl1<m; cl1++){
    for(cl2=0; cl2<m; cl2++){
      for(s=0; s<n_shifts; s++){
        p1 = 0;
        p2 = 0;
        /* inner most dim of dot (matmul) */
        for(k=0; k<r; k++){
          pfik = pf[k*m*n + cl1*n + i];
          pfjk = conj(pf[k*m*n + cl2*n + j]) * shift_phases[s*r + k];
          p1 += real(pfik) * real(pfjk);
          p2 += imag(pfik) * imag(pfjk);
        } /* k */

        xcorr = p1 - p2;
        if(xcorr > best_cl_xcorr){
          best_cl_xcorr = xcorr;
          best_cl1 = cl1;
          best_cl2 = cl2;
        }

        xcorr = p1 + p2;
        if(xcorr > best_cl_xcorr){
          best_cl_xcorr = xcorr;
          best_cl1 = cl1;
          best_cl2 = cl2 + m; /* m is pf.shape[1], which should be n_theta//2 */
        }

      } /* s */
    } /* cl2 */
  }/* cl1 */

  /* update global best for i, j */
  clmatrix[i*n + j] = best_cl1;
  clmatrix[j*n+i] = best_cl2;  /* [j,i] */

} /* build_clmatrix_kernel */

extern "C" __global__
void fbuild_clmatrix_kernel(
    const int n,
    const int m,
    const int r,
    const complex<float>* __restrict__ pf,
    int16_t* const __restrict__ clmatrix,
    const int n_shifts,
    const complex<float>* const __restrict__ shift_phases)
{
  /* n n_img */
  /* m angular componentns, n_theta//2 */
  /* r radial componentns */
  /* (n, m, r) = pf.shape in python (before transpose for CUDA kernel) */

  /* thread index (2d), represents "i" and "j" indices */
  const unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  const unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;

  /* no-op when out of bounds */
  if(i >= n) return;
  if(j >= n) return;
  /* no-op lower triangle */
  if(j <= i) return;

  int k;
  int s;
  int cl1, cl2;
  int best_cl1, best_cl2;
  float xcorr, best_cl_xcorr;
  float p1, p2;
  complex<float> pfik, pfjk;

  best_cl1 = -1;
  best_cl2 = -1;
  best_cl_xcorr = -INFINITY;

  for(cl1=0; cl1<m; cl1++){
    for(cl2=0; cl2<m; cl2++){
      for(s=0; s<n_shifts; s++){
        p1 = 0;
        p2 = 0;
        /* inner most dim of dot (matmul) */
        for(k=0; k<r; k++){
          pfik = pf[k*m*n + cl1*n + i];
          pfjk = conj(pf[k*m*n + cl2*n + j]) * shift_phases[s*r + k];
          p1 += real(pfik) * real(pfjk);
          p2 += imag(pfik) * imag(pfjk);
        } /* k */

        xcorr = p1 - p2;
        if(xcorr > best_cl_xcorr){
          best_cl_xcorr = xcorr;
          best_cl1 = cl1;
          best_cl2 = cl2;
        }

        xcorr = p1 + p2;
        if(xcorr > best_cl_xcorr){
          best_cl_xcorr = xcorr;
          best_cl1 = cl1;
          best_cl2 = cl2 + m; /* m is pf.shape[1], which should be n_theta//2 */
        }

      } /* s */
    } /* cl2 */
  }/* cl1 */

  /* update global best for i, j */
  clmatrix[i*n + j] = best_cl1;
  clmatrix[j*n+i] = best_cl2;  /* [j,i] */

} /* fbuild_clmatrix_kernel */
